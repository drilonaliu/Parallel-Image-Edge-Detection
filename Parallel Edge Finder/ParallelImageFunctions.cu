#include "hip/hip_runtime.h"
#include "ParallelImageFunctions.cuh"
#include "KernelImage.cuh"

Mat parallelFindEges(Mat image) {

	Mat edgedImage(image.rows, image.cols, CV_8U);

	//Size
	int imageSize = image.rows * image.cols;

	//Device pointers 
	uchar* d_image;
	uchar* d_edgedImage;

	//Memory Allocation
	hipMalloc((void**)&d_image, imageSize * sizeof(uchar));
	hipMalloc((void**)&d_edgedImage, imageSize * sizeof(uchar));

	//Memory copy
	hipMemcpy(d_image, image.data, imageSize * sizeof(uchar), hipMemcpyHostToDevice);

	//Launch Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (int)ceil(imageSize / threadsPerBlock);
	int totalThreadsLaunched = threadsPerBlock * blocksPerGrid;
	cudaFindImageEdges << <blocksPerGrid, threadsPerBlock >> > (d_image, d_edgedImage,image.cols,image.rows);

	//Wait For Cuda
	hipDeviceSynchronize();

	//Copy Results back
	hipMemcpy(edgedImage.data, d_edgedImage, imageSize* sizeof(uchar), hipMemcpyDeviceToHost);

	//Free device memory
	hipFree(d_image);
	hipFree(d_edgedImage);

	return edgedImage;
}